#include "hip/hip_runtime.h"
/*
* MIT License
*
* Copyright(c) 2010 Denis Gladkov
*
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files(the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions :
*
* The above copyright notice and this permission notice shall be included in all
* copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/

#include "random/mtwister.cuh"
#include "kernels.h"
#include "common.h"

__global__	void	TestMersenneTwisterGPU(int*	data, int count)
{
	const	int		tid = blockDim.x*blockIdx.x + threadIdx.x;
	const	int	numThreads = blockDim.x*gridDim.x;

	cudahelp::rand::mt_state_t	state = cudahelp::rand::GetMTState(tid);
	cudahelp::rand::mt_struct_stripped_t	config = cudahelp::rand::GetMTConfig(tid);

	for(int idx = tid; idx < count; idx += numThreads)
		data[idx] = cudahelp::rand::MTwisterRndInt(state, config);

	cudahelp::rand::SaveMTState(tid, state);
}       

__global__	void	TestMersenneTwisterClassGPU(int*	data, int count)
{
	int		idx = blockDim.x*blockIdx.x + threadIdx.x;
	const	int	numThreads = blockDim.x*gridDim.x;

	cudahelp::rand::MTGenerator	gen(idx);

	for(; idx < count; idx += numThreads)
		data[idx] = gen.GetInt();
}

void	TestMersenneTwister(int*	data, int count, int rngs)
{
	int	numThreads = 256;
	int	numBlocks = cudahelp::GetNumberOfBlocks(numThreads, rngs);

	TestMersenneTwisterGPU<<<numBlocks, numThreads>>>(data, count);

	hipDeviceSynchronize();

	cudahelp::CheckCUDAError("TestMersenneTwisterGPU");

}

void	TestMersenneTwisterClass(int*	data, int count, int rngs)
{
	int	numThreads = 256;
	int	numBlocks = cudahelp::GetNumberOfBlocks(numThreads, rngs);

	TestMersenneTwisterClassGPU<<<numBlocks, numThreads>>>(data, count);

	hipDeviceSynchronize();

	cudahelp::CheckCUDAError("TestMersenneTwisterClassGPU");
}
